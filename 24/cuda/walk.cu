#include "hip/hip_runtime.h"
#define HIPCUB_STDERR
#include <hip/hip_cooperative_groups.h>
#include <cub/util_debug.cuh>
#include <cuda/std/barrier>
#include <experimental/mdspan>
#include <string>
#include "parse.h"

// modified from https://stackoverflow.com/a/14038590
#include <assert.h>
#define cdpErrchk(ans) \
  { cdpAssert((ans), __FILE__, __LINE__); }
__device__ void cdpAssert(
    hipError_t code, const char* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    printf(
        "%s:%d GPU kernel assert %d: %s \n", file, line, code,
        hipGetErrorString(code));
    if (abort)
      assert(0);
  }
}

using myspan = std::experimental::mdspan<
    int, std::experimental::extents<
             size_t, std::experimental::dynamic_extent,
             std::experimental::dynamic_extent>>;

__device__ dim3 mapsize;

__device__ mysizet x_begin() {
  return threadIdx.x * (mapsize.x / blockDim.x + !!(mapsize.x % blockDim.x));
}
__device__ mysizet x_end() {
  return min(
      (threadIdx.x + 1) * (mapsize.x / blockDim.x + !!(mapsize.x % blockDim.x)),
      mapsize.x);
}

__device__ mysizet y_begin() {
  return threadIdx.y * (mapsize.y / blockDim.y + !!(mapsize.y % blockDim.y));
}
__device__ mysizet y_end() {
  return min(
      (threadIdx.y + 1) * (mapsize.y / blockDim.y + !!(mapsize.y % blockDim.y)),
      mapsize.y);
}

__device__ mysizet upstorm_y(mysizet y_now, mysizet round) {
  return (y_now + round) % mapsize.y;
}

__device__ mysizet leftstorm_x(mysizet x_now, mysizet round) {
  return (x_now + round) % mapsize.x;
}

__device__ mysizet rightstorm_x(mysizet x_now, mysizet round) {
  while (round > x_now && round >= mapsize.x) {
    round -= mapsize.x;
  }

  if (round <= x_now)
    return x_now - round;
  return (x_now + mapsize.x) - round;
}

__device__ mysizet downstorm_y(mysizet y_now, mysizet round) {
  while (round > y_now && round >= mapsize.y) {
    round -= mapsize.y;
  }

  if (round <= y_now)
    return y_now - round;
  return (y_now + mapsize.y) - round;
}

__device__ cuda::barrier<cuda::thread_scope_block>::arrival_token print(
    myspan& storm_left, myspan& storm_right, myspan& storm_up,
    myspan& storm_down, myspan& exploration, int& round,
    cuda::barrier<cuda::thread_scope_block>& barrier,
    cuda::barrier<cuda::thread_scope_block>::arrival_token&& token) {
  barrier.wait(std::move(token));
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    for (int x = 0; x < mapsize.x + 2; ++x) {
      printf("#");
    }
    printf("\n");
    for (int y = 0; y < mapsize.y; ++y) {
      printf("#");
      for (int x = 0; x < mapsize.x; ++x) {
        char to_be_put = '.';
        if (exploration(y, x))
          to_be_put = 'E';
        if (storm_left(y, leftstorm_x(x, round)))
          to_be_put = '<';
        if (storm_right(y, rightstorm_x(x, round)))
          to_be_put = '>';
        if (storm_up(upstorm_y(y, round), x))
          to_be_put = '^';
        if (storm_down(downstorm_y(y, round), x))
          to_be_put = 'v';
        if (auto sum = storm_left(y, leftstorm_x(x, round)) +
                       storm_right(y, rightstorm_x(x, round)) +
                       storm_up(upstorm_y(y, round), x) +
                       storm_down(downstorm_y(y, round), x);
            sum > 1)
          to_be_put = sum + '0';
        printf("%c", to_be_put);
      }
      printf("#\n");
    }
    for (int x = 0; x < mapsize.x + 2; ++x) {
      printf("#");
    }
    printf("\n\n\n\n");
  }
}

__device__ void explore(
    myspan const& prev, myspan& next, myspan& storm_left, myspan& storm_right,
    myspan& storm_up, myspan& storm_down, int next_round,
    cuda::barrier<cuda::thread_scope_block>& barrier) {
  for (auto y = y_begin(); y < y_end(); ++y) {
    for (auto x = x_begin(); x < x_end(); ++x) {
      next(y, x) = 0;
      if (y == 0 && x == 0) {
        next(y, x) = 1;
      }
      if (prev(y, x)) {
        next(y, x) = 1;
      }
      if (y > 0 && prev(y - 1, x)) {
        next(y, x) = 1;
      }
      if (y < mapsize.y - 1 && prev(y + 1, x)) {
        next(y, x) = 1;
      }
      if (x > 0 && prev(y, x - 1)) {
        next(y, x) = 1;
      }
      if (x < mapsize.x - 1 && prev(y, x + 1)) {
        next(y, x) = 1;
      }
      if (auto sum = storm_left(y, leftstorm_x(x, next_round)) +
                     storm_right(y, rightstorm_x(x, next_round)) +
                     storm_up(upstorm_y(y, next_round), x) +
                     storm_down(downstorm_y(y, next_round), x);
          sum > 0) {
        next(y, x) = 0;
      }
    }
  }

  barrier.arrive_and_wait();
}

__global__ void proceed(int* map_arg, int* X, int* Y) {
  __shared__ myspan map;
  __shared__ myspan storm_left;
  __shared__ myspan storm_right;
  __shared__ myspan storm_up;
  __shared__ myspan storm_down;
  __shared__ myspan exploration_a;
  __shared__ myspan exploration_b;

  auto block = cooperative_groups::this_thread_block();
  __shared__ cuda::barrier<cuda::thread_scope_block> barrier;
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    mapsize.x = *X;
    mapsize.y = *Y;
    auto storage_size = mapsize.y * mapsize.x;
    auto block_size = blockDim.x * blockDim.y;
    init(&barrier, block_size);
    map = myspan(map_arg, mapsize.y, mapsize.x);
    int* store;
    cdpErrchk(hipMalloc(&store, storage_size * sizeof(int)));
    storm_left = myspan(store, mapsize.y, mapsize.x);
    cdpErrchk(hipMalloc(&store, storage_size * sizeof(int)));
    storm_right = myspan(store, mapsize.y, mapsize.x);
    cdpErrchk(hipMalloc(&store, storage_size * sizeof(int)));
    storm_up = myspan(store, mapsize.y, mapsize.x);
    cdpErrchk(hipMalloc(&store, storage_size * sizeof(int)));
    storm_down = myspan(store, mapsize.y, mapsize.x);
    cdpErrchk(hipMalloc(&store, storage_size * sizeof(int)));
    exploration_a = myspan(store, mapsize.y, mapsize.x);
    cdpErrchk(hipMalloc(&store, storage_size * sizeof(int)));
    exploration_b = myspan(store, mapsize.y, mapsize.x);
  }
  block.sync();
  cdpErrchk(hipPeekAtLastError());

  for (auto y = y_begin(); y < y_end(); ++y) {
    for (auto x = x_begin(); x < x_end(); ++x) {
      storm_left(y, x) = 0;
      storm_right(y, x) = 0;
      storm_up(y, x) = 0;
      storm_down(y, x) = 0;
      if (map(y, x) == '<') {
        storm_left(y, x) = 1;
      } else if (map(y, x) == '^') {
        storm_up(y, x) = 1;
      } else if (map(y, x) == '>') {
        storm_right(y, x) = 1;
      } else if (map(y, x) == 'v') {
        storm_down(y, x) = 1;
      }
      exploration_a(y, x) = 0;
    }
  }
  barrier.arrive_and_wait();

  auto round = 0;
  for (; round <= 2000; ++round) {
    myspan& prev = (round % 2 == 0) ? exploration_a : exploration_b;
    myspan& next = (round % 2 == 0) ? exploration_b : exploration_a;
    // auto t = barrier.arrive();
    // print(
    //     storm_left, storm_right, storm_up, storm_down, prev, round, barrier,
    //     std::move(t));
    explore(
        prev, next, storm_left, storm_right, storm_up, storm_down, round + 1,
        barrier);

    if (prev(mapsize.y -1, mapsize.x -1)) break;
  }

  if (threadIdx.x == 0 && threadIdx.y == 0) {
    printf("Reached goal after %d rounds\n", round+1);
    hipFree(storm_up.data_handle());
    hipFree(storm_down.data_handle());
    hipFree(storm_left.data_handle());
    hipFree(storm_right.data_handle());
    hipFree(exploration_a.data_handle());
    hipFree(exploration_b.data_handle());
  }
}

__device__ void p() {
  printf(
      "(%d, %d) out of %d x %d\n", threadIdx.x, threadIdx.y, blockDim.x,
      blockDim.y);
}

__global__ void test() {
  p();
}

int main(int, char** argv) {
  int* k;
  int* X;
  int* Y;

  auto i = read(argv[1]);
  printf("there are %d rows and %d cols\n", std::get<1>(i), std::get<2>(i));

  CubDebugExit(hipMallocManaged(&X, sizeof(int)));
  CubDebugExit(hipMallocManaged(&Y, sizeof(int)));
  *X = std::get<2>(i);
  *Y = std::get<1>(i);
  auto mapsize = (*X) * (*Y);

  CubDebugExit(hipMallocManaged(&k, mapsize * sizeof(int)));
  for (int ii = 0; ii < mapsize; ++ii) {
    k[ii] = std::get<0>(i)[ii];
  }

  proceed<<<1, {32, 32}>>>(k, X, Y);
  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
  CubDebugExit(hipPeekAtLastError());
}
