#include <cstdio>
#include <cub/util_debug.cuh>
#include <thrust/device_vector.h>
#include "global_funs.cuh"
#include "io_format.h"

int main(int, char**) {
  auto input = read("../input.txt");

  thrust::device_vector<int> d_x = input.first;
  thrust::device_vector<int> d_y = input.second;

  thrust::device_vector<int> d_px(input.first.size());
  thrust::device_vector<int> d_py(input.first.size());
  thrust::device_vector<int> d_go(input.first.size());

  int* N_elves;
  int* rm4;
  int* res;
  CubDebugExit(hipMallocManaged(&N_elves, sizeof(int)));
  CubDebugExit(hipMallocManaged(&rm4, sizeof(int)));
  CubDebugExit(hipMallocManaged(&res, sizeof(int)));
  *N_elves = input.first.size();
  *rm4 = 0;

  for (int r = 0; r < 10; ++r) {
    *rm4 = r % 4;

    do_round<<<1, 1>>>(
        N_elves, thrust::raw_pointer_cast(d_x.data()),
        thrust::raw_pointer_cast(d_y.data()),
        thrust::raw_pointer_cast(d_go.data()),
        thrust::raw_pointer_cast(d_px.data()),
        thrust::raw_pointer_cast(d_py.data()), rm4);
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());
    CubDebugExit(hipPeekAtLastError());
  }
  empty_spaces<<<1, 1>>>(
      thrust::raw_pointer_cast(d_x.data()),
      thrust::raw_pointer_cast(d_y.data()), N_elves, res);
  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
  CubDebugExit(hipPeekAtLastError());

  printf("In the minimal map there ought to be %d empty cells\n", *res);

  CubDebugExit(hipFree(N_elves));
  CubDebugExit(hipFree(rm4));
}
