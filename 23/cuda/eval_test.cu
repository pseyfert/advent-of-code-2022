#include "global_funs.cuh"
#include "io_format.h"

#include <cub/util_debug.cuh>
#include <thrust/device_vector.h>

#include <gtest/gtest.h>

TEST(example, count_empty) {
  auto input = read("../end_example.txt");

  thrust::device_vector<int> d_x = input.first;
  thrust::device_vector<int> d_y = input.second;

  int* N_elves;
  int* res;
  CubDebugExit(hipMallocManaged(&N_elves, sizeof(int)));
  CubDebugExit(hipMallocManaged(&res, sizeof(int)));
  *N_elves = input.first.size();

  empty_spaces<<<1, 1>>>(
      thrust::raw_pointer_cast(d_x.data()),
      thrust::raw_pointer_cast(d_y.data()), N_elves, res);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
  CubDebugExit(hipPeekAtLastError());

  EXPECT_EQ(*res, 110);

  CubDebugExit(hipFree(N_elves));
  CubDebugExit(hipFree(res));
}
