#include "global_funs.cuh"
#include "io_format.h"

#include <cub/util_debug.cuh>
#include <stdio.h>
#include <thrust/device_vector.h>

int main() {
  auto input = read("../end_example.txt");

  thrust::device_vector<int> d_x = input.first;
  thrust::device_vector<int> d_y = input.second;

  int* N_elves;
  int* res;
  CubDebugExit(hipMallocManaged(&N_elves, sizeof(int)));
  CubDebugExit(hipMallocManaged(&res, sizeof(int)));
  *N_elves = input.first.size();

  empty_spaces<<<1, 1>>>(
      thrust::raw_pointer_cast(d_x.data()),
      thrust::raw_pointer_cast(d_y.data()), N_elves, res);

  CubDebugExit(hipPeekAtLastError());
  CubDebugExit(hipDeviceSynchronize());
  CubDebugExit(hipPeekAtLastError());

  printf("in the example there are %d empty cells\n", *res);

  CubDebugExit(hipFree(N_elves));
  CubDebugExit(hipFree(res));

  return 0;
}
